#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <cusp/complex.h>

// borrow petsc types
typedef double PetscScalar;
typedef cusp::complex<PetscScalar> PetscComplex;
typedef int PetscInt;

// FIXME global var used to pass array of data
int const MaxSpecies = 2;
int const MaxSize    = 2*MaxSpecies ;
int const tol        = 1.E-4;
int const upperbound = 100;


/*
  filter operation
   \hat{Y} =  Y/D
*/
__device__ 
void  signalfilter(int Necho,PetscComplex *y,int Nspecies,PetscComplex beta[])
{
     for (int iii=0;iii<Necho;iii++) 
        for (int jjj=0;jjj<Nspecies;jjj++) 
           y[iii] = y[iii] - beta[jjj] *y[iii-(jjj+1)];
}
__device__
void WriteSolution(PetscComplex a[][MaxSize],int n,PetscComplex *x)
{
   int j,k;

   for (j=0;j<n;j++) {
      for (k=0;k<n+1;k++) {
         printf("%d %d %12.5e %12.5e ",k,j,a[k][j].real(),a[k][j].imag());
      }
      printf(" | %d  %12.5e %12.5e \n",j,x[j].real(),x[j].imag());
   }
   printf("\n");
}
/*
 * Device code
   Solve a system of n equations in n unknowns using Gaussian Elimination
   Solve an equation in matrix form Ax = b
   The 2D array a is the matrix A with an additional column b.
   This is often written (A:b)

   TODO notice that the first index is the largest dimension
   A0,0    A1,0    A2,0    ....  An-1,0     b0
   A0,1    A1,1    A2,1    ....  An-1,1     b1
   A0,2    A1,2    A2,2    ....  An-1,2     b2
   :       :       :             :          :
   :       :       :             :          :
   A0,n-1  A1,n-1  A2,n-1  ....  An-1,n-1   bn-1
 */
__device__ 
void  GSolve(PetscComplex a[][MaxSize],int n,PetscComplex x[])
{
   int i,j,k; //,maxrow;
   PetscComplex tmp;

   for (i=0;i<n;i++) {

      ///* Find the row with the largest first value */
      //maxrow = i;
      //for (j=i+1;j<n;j++) {
      //   if (ABS(a[i][j]) > ABS(a[i][maxrow]))
      //      maxrow = j;
      //}

      ///* Swap the maxrow and ith row */
      //for (k=i;k<n+1;k++) {
      //   tmp = a[k][i];
      //   a[k][i] = a[k][maxrow];
      //   a[k][maxrow] = tmp;
      //}

      ///* Singular matrix? */
      //if (ABS(a[i][i]) < EPS)
      //   return(FALSE);

      /* Eliminate the ith element of the jth row */
      for (j=i+1;j<n;j++) {
         for (k=n;k>=i;k--) {
            a[k][j] -= a[k][i] * a[i][j] / a[i][i];
         }
      }
   }

   /* Do the back substitution */
   for (j=n-1;j>=0;j--) {
      tmp = 0;
      for (k=j+1;k<n;k++)
         tmp += a[k][j] * x[k];
      x[j] = (a[n][j] - tmp) / a[j][j];
   }

   return;
}

/*************QR Root Solve*************/
__device__
PetscComplex dotprod(PetscComplex *vec1, PetscComplex *vec2, int nDim)
{
	PetscComplex x = 0;
	PetscComplex tmp = 0;
	for (int i = 0; i < nDim; ++i)
	{
		tmp.real(vec1[i].real());
		tmp.imag(-vec1[i].imag());
		x += tmp * vec2[i];
	}
	return x;
}

__device__
PetscComplex* matmult(PetscComplex *mat1, PetscComplex *mat2, int nDim)
{
	PetscComplex *x = new PetscComplex[nDim * nDim];
	for (int i = 0; i < nDim * nDim; ++i)
		x[i] = 0;
	for (int k = 0; k < nDim; ++k)
		for (int j = 0; j < nDim; ++j)
			for (int i = 0; i < nDim; ++i)
				x[j + k * nDim] += mat1[j + i * nDim] * mat2[i + k * nDim];
	return x;
}

__device__
PetscComplex l2norm(PetscComplex *vec, int nDim)
{
	PetscComplex x = 0;
	for (int i = 0; i < nDim; ++i)
		x += vec[i].real() * vec[i].real() + vec[i].imag() * vec[i].imag();
	x = sqrt(x);
	return x;
}

__device__
void make_comp_mat(PetscComplex *polynomial, PetscComplex *companion, int nDim)
{
	for (int i = 0; i < nDim * nDim; ++i)
		companion[i] = 0;
	for (int i = 0; i < nDim; ++i)
		companion[i * nDim] = -polynomial[i + 1] / polynomial[0];
	for (int i = 0; i < nDim - 1; ++i)
		companion[i * nDim + i + 1] = 1;
}

__device__
void select_diag(PetscComplex *vector, PetscComplex *matrix, int nDim)
{
	for (int i = 0; i < nDim; ++i)
		vector[i] = matrix[i * nDim + i];
}

__device__
void modified_gram_schmidt(PetscComplex *a, PetscComplex *Q, PetscComplex *R, int nDim)
{
	PetscComplex *u = new PetscComplex[nDim];
	PetscComplex *v = new PetscComplex[nDim];
	PetscComplex prj = 0;
	PetscComplex l2 = 0;
	for (int i = 0; i < nDim * nDim; ++i)
		Q[i] = R[i] = 0;
	for (int i = 0; i < nDim; ++i)
		u[i] = v[i] = 0;

	for (int k = 0; k < nDim; ++k)
	{
		for (int i = 0; i < nDim; ++i)
			u[i] = a[i + k * nDim];
		for (int j = 0; j < k; ++j)
		{
			for (int i = 0; i < nDim; ++i)
				v[i] = Q[i + j * nDim];
			prj = dotprod(v, u, nDim);
			for (int i = 0; i < nDim; ++i)
				u[i] -= prj * v[i];
		}
		l2 = l2norm(u, nDim);
		for (int i = 0; i < nDim; ++i)
			Q[i + k * nDim] = u[i] / l2;
		for (int j = k; j < nDim; ++j)
		{
			for (int i = 0; i < nDim; ++i)
			{
				u[i] = a[i + j * nDim];
				v[i] = Q[i + k * nDim];
			}
			R[k + j * nDim] = dotprod(u, v, nDim);
		}
	}

	delete[] u;
	delete[] v;
}

__device__
void roots(
	PetscComplex *polynomial,
	PetscComplex *root,
	int nDim_in,
	double tolerance,
	int upperbound)
{
	int nDim = nDim_in - 1;
	PetscComplex *a = new PetscComplex[nDim * nDim];
	PetscComplex *Q = new PetscComplex[nDim * nDim];
	PetscComplex *R = new PetscComplex[nDim * nDim];
	int nTol = 0;
	for (int i = 0; i < nDim; ++i)
		root[i] = 0;

	make_comp_mat(polynomial, a, nDim);

	for (int k = 0; k < upperbound; ++k)
	{
		modified_gram_schmidt(a, Q, R, nDim);
		a = matmult(R, Q, nDim);
		nTol = 0;
		for (int j = 0; j < nDim; ++j)
			for (int i = 0; i < nDim; ++i) {
				if (i > j && sqrt(a[i + j * nDim].real() * a[i + j * nDim].real() + 
					a[i + j * nDim].imag() * a[i + j * nDim].imag()) > tolerance) 
					++nTol; }
		if (nTol == 0) break;
	}

	select_diag(root, a, nDim);

	delete[] a;
	delete[] Q;
	delete[] R;
}
/*************End QR Root Solve*************/

/*
 * Device code
 */
__global__ 
void StmcbKernel(
         const double* d_RealDataArray,
         const double* d_ImagDataArray,
               double* d_Ppm,
               double* d_T2star,
               double* d_Amplitude,
               double* d_Phase,
         double const EchoSpacing,
         double const ImagingFreq,
         double const ThresholdSignal,
         int const Necho,
         int const Nspecies,
         int const Npixel,
        const int debugthread   , 
        const int displaythread , 
        const int debugverbose  ) 
{
    /*
      grid stride loop design pattern, 1-d grid
      http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
         - By using a loop, you can support any problem size even if it exceeds the largest grid size your CUDA device supports. Moreover, you can limit the number of blocks you use to tune performance.
    */
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; 
         idx < Npixel;
         idx += blockDim.x * gridDim.x) 
      {
        /* define temporary data structures in register memory */
        int iii,jjj,kkk;
        int const KernelMaxEcho = 16;
        // array for original data
        PetscComplex  dataworkfull[KernelMaxEcho+MaxSpecies];
        PetscComplex  sysinputfull[KernelMaxEcho+MaxSpecies];
        // initialize
        for(iii = 0; iii< KernelMaxEcho+MaxSpecies; iii++)
         {
           dataworkfull[iii] = 0.0;
           sysinputfull[iii] = 0.0;
         }
        // setup pointer to allow negative index during assembly
        PetscComplex  *datawork = &dataworkfull[MaxSpecies];
        PetscComplex  *sysinput = &sysinputfull[MaxSpecies];

        // storage for augmented matrix 
        PetscComplex  wrkMatrix[MaxSize+1][MaxSize];
        // storage for solution = [beta_1 ... beta_Nspecies alpha_0 ... alpha_{Nspecies-1}]
        PetscComplex  slnVector[MaxSize];
        
        if (idx == debugthread ) printf("idx=%d Necho=%d Nspecies=%d Npixel=%d\n",idx,Necho, Nspecies, Npixel);
              
        if (Necho > KernelMaxEcho)  // error check
          printf("Necho=%d > %d not supported \n",Necho,KernelMaxEcho);
        else if (Nspecies > MaxSpecies)  // error check
          printf("NSpecies=%d > %d not supported \n",Nspecies,MaxSpecies);
        else 
         {
           /* Copy Global data to register memory (ie Opencl Private) */
           double signalmagnitude = 0.0;
           for(iii = 0; iii< Necho; iii++)
            {
              datawork[iii] = PetscComplex(d_RealDataArray[idx * Necho+iii],d_ImagDataArray[idx * Necho+iii]);
              signalmagnitude  = signalmagnitude  + cusp::abs(datawork[iii]) ;
              sysinput[iii] = 0.0;
              if (idx == debugthread ) printf("idx=%d datawork[%d]=(%f,%f) sysinput[%d]=(%f,%f), mag = %f\n",idx,iii,datawork[iii].real(),datawork[iii].imag(),iii,sysinput[iii].real(),sysinput[iii].imag(),signalmagnitude  );
            }
           // only compute for sufficient signal
           if(signalmagnitude > ThresholdSignal)
            {
              // system input is deltat function
              sysinput[0] = 1.0;

              /* initialize */
              for(iii = 0; iii< Nspecies+1; iii++)
                  for(jjj = 0; jjj< Nspecies; jjj++)
                    wrkMatrix[iii][jjj] = 0.0;

              /* Build Matrix and RHS for prony solve  */
              for(iii = 0; iii< Nspecies; iii++)
               {
                for(jjj = 0; jjj< Nspecies; jjj++)
                  {
                    for(kkk = Nspecies; kkk< Necho; kkk++)
                      {
                       // TODO Notice Gauss Solver is Row MAJOR
                       //wrkMatrix[iii][jjj] = wrkMatrix[iii][jjj] + cusp::conj(datawork[kkk-iii-1]) * datawork[kkk-jjj-1];
                       wrkMatrix[jjj][iii] = wrkMatrix[jjj][iii] + cusp::conj(datawork[kkk-iii-1]) * datawork[kkk-jjj-1];
                      }
                  }
                for(kkk = Nspecies; kkk< Necho; kkk++)
                  {
                    wrkMatrix[Nspecies][iii] = wrkMatrix[Nspecies][iii]  - cusp::conj(datawork[kkk-iii-1]) * datawork[kkk];
                    //if (idx == debugthread ) printf("idx=%d %d, %d,%d \n",idx,iii,kkk-iii-1,kkk);
                  }
               }

              // initialize solution
              for(iii = 0; iii< 2*Nspecies; iii++) slnVector[iii] = 0.0;

              /* solve prony linear system */
              if (idx == debugthread ) printf("Prony:\n");
              if (idx == debugthread ) WriteSolution(wrkMatrix,Nspecies,slnVector);
              GSolve(wrkMatrix,Nspecies,slnVector);
              if (idx == debugthread ) WriteSolution(wrkMatrix,Nspecies,slnVector);


              // buffer for stmcb iteration
              PetscComplex  dataworkstmcb[KernelMaxEcho+MaxSpecies];
              PetscComplex  sysinputstmcb[KernelMaxEcho+MaxSpecies];

              /* steiglitz iteration */
              for(int isteig = 0 ; isteig <5 ; isteig++)
                {
                   // initialize - restore pre-filter data
                   for(iii = 0; iii< KernelMaxEcho+MaxSpecies; iii++)
                    {
                      dataworkstmcb[iii] = dataworkfull[iii] ;
                      sysinputstmcb[iii] = sysinputfull[iii] ;
                    }
                   // initialize
                   for(iii = 0; iii< 2* Nspecies+1; iii++)
                       for(jjj = 0; jjj< 2* Nspecies; jjj++)
                         wrkMatrix[iii][jjj] = 0.0;
                   // FIXME - bad practice - difficult to follow
                   // setup pointer to allow negative index during assembly
                   datawork = &dataworkstmcb[MaxSpecies];
                   sysinput = &sysinputstmcb[MaxSpecies];

                     if (idx == debugthread )
                       {
                       PetscComplex tmpcheck=(datawork[2]-slnVector[0]*(datawork[1]-slnVector[0]*datawork[0])-slnVector[1]*datawork[0]);
                       printf("expected filter w[2]=(%12.5e,%12.5e) beta[0]=(%12.5e,%12.5e)\n",tmpcheck.real(),tmpcheck.imag(),slnVector[0].real(), slnVector[0].imag() );
                       }
                  signalfilter(Necho,datawork,Nspecies,slnVector);
                  signalfilter(Necho,sysinput,Nspecies,slnVector);
                  for(iii = 0; iii< Necho; iii++)
                     if (idx == debugverbose ) printf("idx=%d datawork[%d]=(%12.5e,%12.5e) sysinput[%d]=(%12.5e,%12.5e)\n",idx,iii,datawork[iii].real(),datawork[iii].imag(),iii,sysinput[iii].real(),sysinput[iii].imag());
                  /* Build Matrix and RHS for steiglitz  solve  */
                  for(iii = 0; iii< Nspecies; iii++)
                   {
                    // matrix
                    for(jjj = 0; jjj< Nspecies; jjj++)
                      {
                        for(kkk = 0; kkk< Necho; kkk++)
                          {
                           // TODO Notice Gauss Solver is Row MAJOR
                           wrkMatrix[jjj         ][iii         ] = wrkMatrix[jjj         ][iii         ] + cusp::conj(-datawork[kkk-iii-1]) *-datawork[kkk-jjj-1];
                           wrkMatrix[jjj+Nspecies][iii         ] = wrkMatrix[jjj+Nspecies][iii         ] + cusp::conj(-datawork[kkk-iii-1]) * sysinput[kkk-jjj  ];
                           wrkMatrix[jjj         ][iii+Nspecies] = wrkMatrix[jjj         ][iii+Nspecies] + cusp::conj( sysinput[kkk-iii  ]) *-datawork[kkk-jjj-1];
                           wrkMatrix[jjj+Nspecies][iii+Nspecies] = wrkMatrix[jjj+Nspecies][iii+Nspecies] + cusp::conj( sysinput[kkk-iii  ]) * sysinput[kkk-jjj  ];
                          }
                      }
                    // rhs
                    for(kkk = 0; kkk< Necho; kkk++)
                      {
                        wrkMatrix[2*Nspecies][iii         ] = wrkMatrix[2*Nspecies][iii         ] + cusp::conj(-datawork[kkk-iii-1]) * datawork[kkk];
                        // offest by Nspecies to for the signal input
                        wrkMatrix[2*Nspecies][iii+Nspecies] = wrkMatrix[2*Nspecies][iii+Nspecies] + cusp::conj( sysinput[kkk-iii  ]) * datawork[kkk];
                        //if (idx == debugthread ) printf("idx=%d %d, %d,%d \n",idx,iii,kkk-iii-1,kkk);
                      }
                   }

                  /* solve */
                  if (idx == debugverbose ) WriteSolution(wrkMatrix,2*Nspecies,slnVector);
                  GSolve(wrkMatrix,2*Nspecies,slnVector);
                  if (idx == debugthread ) WriteSolution(wrkMatrix,2*Nspecies,slnVector);
                }

              // analytic 1 peak 
              PetscComplex  Lambda[MaxSpecies];
              PetscComplex  amplitude[MaxSpecies];
              // initialize amplitude
              for(iii = 0; iii< Nspecies; iii++) amplitude[iii] = 0.0;

              if ( Nspecies ==  1 )  
                {
                  /* compute roots */
                  Lambda[0] = - slnVector[0] ;
                  /* compute amplitude from residue */
                  amplitude[0] = (slnVector[1])/(slnVector[0]*Lambda[0]);
                }
              // analytic 2 peak 
              else if ( Nspecies ==  2 )  
                {
                  if (idx == displaythread ) printf("idx=%d alpha[0]=(%f,%f) alpha[1]=(%f,%f) beta[1]=(%f,%f) beta[2]=(%f,%f)\n",idx,
                                         slnVector[2].real(),slnVector[2].imag(),
                                         slnVector[3].real(),slnVector[3].imag(),
                                         slnVector[0].real(),slnVector[0].imag(),
                                         slnVector[1].real(),slnVector[1].imag()
                                       );
                  /* compute roots */
                  Lambda[0] = 0.5 * ( slnVector[0] + sqrt(slnVector[0]*slnVector[0] + 4.0 * slnVector[1]) ); 
                  Lambda[1] = 0.5 * ( slnVector[0] - sqrt(slnVector[0]*slnVector[0] + 4.0 * slnVector[1]) ); 
               
                  /* compute amplitude from initial conditions*/
                  wrkMatrix[0][0] = 1;
                  wrkMatrix[1][0] = 1;
                  wrkMatrix[0][1] = Lambda[0]+slnVector[0];
                  wrkMatrix[1][1] = Lambda[1]+slnVector[0];
                  wrkMatrix[2][0] = slnVector[2];
                  wrkMatrix[2][1] = slnVector[3];

                  if (idx == debugthread ) WriteSolution(wrkMatrix,Nspecies,amplitude);
                  GSolve(wrkMatrix,2*Nspecies,amplitude);
                  if (idx == debugthread ) WriteSolution(wrkMatrix,Nspecies,amplitude);
                  if (idx == displaythread ) printf("idx=%d Lambda[0]=(%12.5e,%12.5e) Lambda[1]=(%12.5e,%12.5e) \n",idx,Lambda[0].real(),Lambda[0].imag(),Lambda[1].real(),Lambda[1].imag());
                  if (idx == displaythread ) printf("idx=%d Amp[0]=(%12.5e,%12.5e) Amp[1]=(%12.5e,%12.5e) \n",idx,amplitude[0].real(),amplitude[0].imag(),amplitude[1].real(),amplitude[1].imag());

                  /* compute amplitude from residue */
                  /* FIXME: extend to multiple species */
                  for(iii = 0; iii< Nspecies; iii++)
                    amplitude[iii] = (slnVector[2]+slnVector[3]*Lambda[iii])/(2.0*slnVector[0]+slnVector[1]*Lambda[iii])/Lambda[iii];

                }
              // TODO Npeaks > 2 needed
              else if (Nspecies > MaxSpecies)  // error check
               	roots(slnVector, Lambda, Nspecies+1, tol, upperbound);

              // compute amplitudes, frequency, and t2star
              for(iii = 0; iii< Nspecies; iii++)
                {
                  PetscComplex  logroot = cusp::log(Lambda[iii]);
                  d_Ppm[      idx*Nspecies+iii] = logroot.imag()/2.0/M_PI/(EchoSpacing*ImagingFreq) * 1.e-3;
                  d_T2star[   idx*Nspecies+iii] = -EchoSpacing/logroot.real() ;

                  d_Amplitude[idx*Nspecies+iii] = cusp::abs(amplitude[iii]) ;
                  d_Phase[    idx*Nspecies+iii] = cusp::arg(amplitude[iii]) ;
                  
                  if (idx == displaythread ) printf("idx=%d ppm[%d]=%12.5e t2star[%d]=%12.5e amplitude[%d]=%12.5e phase[%d]=%12.5e  \n",idx,iii,d_Ppm[idx*Nspecies+iii], iii,d_T2star[   idx*Nspecies+iii], iii,d_Amplitude[idx*Nspecies+iii], iii,d_Phase[    idx*Nspecies+iii] );
                }

            }
           else
            {
              // return default values for no signal
              for(iii = 0; iii< Nspecies; iii++)
                {
                  d_Ppm[      idx*Nspecies+iii] = 0.0;
                  d_T2star[   idx*Nspecies+iii] = 0.0;
                  d_Amplitude[idx*Nspecies+iii] = 0.0;
                  d_Phase[    idx*Nspecies+iii] = 0.0;
                }
            }
         } 
      } // end grid stride loop design pattern, 1-d grid
}


